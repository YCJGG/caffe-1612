#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/inflationX_layer.hpp"
#include "caffe/util/math_functions.hpp"

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

namespace caffe {

template <typename Dtype>
__global__ void InflateForwardGPU(const int nthreads,
          const Dtype* bottom_data, const int bottom_height, const int bottom_width, 
          Dtype *top_data, const int top_height, const int top_width, 
          const float factor, Dtype *factor_diff_matrix, const int margin,
          const float factor_bg_mask=1, const float bg_mask_weight=1, const Dtype* label=NULL) {
           
    const float anchor_y = 0; //(height - 1) / 2.0;
    const float anchor_x = 0; //(width - 1) / 2.0;
    
    const float normalizer = margin * margin * margin * margin;
          
    CUDA_KERNEL_LOOP(index, nthreads) {
        
        // index refers to to top_data
        const int y_t = index / top_width;
        const int x_t = index % top_width;
        
        // coordinate on target map
        const int idx_t = y_t * top_width + x_t;
            
        top_data[idx_t] = 0;
        factor_diff_matrix[idx_t] = 0;
            
        float y_s = y_t / factor;
        float x_s = x_t / factor;
            
        for (int n = MAX(floor(y_s - margin) + 1, 0); n < MIN(y_s + margin, bottom_height); n++) {
            for (int m = MAX(floor(x_s - margin) + 1, 0); m < MIN(x_s + margin, bottom_width); m++) {
             
                top_data[idx_t] += bottom_data[n * bottom_width + m] * (margin - abs(x_s - m)) * (margin - abs(y_s - n));
                    
                factor_diff_matrix[idx_t] += bottom_data[n * bottom_width + m] 
                                             * ((2 * (m >= x_s) - 1) * (margin - abs(y_s - n)) * (-(x_s - anchor_x) / factor)
                                               +(2 * (n >= y_s) - 1) * (margin - abs(x_s - m)) * (-(y_s - anchor_y) / factor));
		// when using background mask
		if (label!= NULL && label[int(round(idx_t*factor_bg_mask))] == 0)
		{
		    factor_diff_matrix[idx_t] *= bg_mask_weight;
		}
		if (label!= NULL && label[int(round(idx_t*factor_bg_mask))] == 255)
		{
		    factor_diff_matrix[idx_t] = 0;
		}
            }
        }
        // normalize
        top_data[idx_t] /= normalizer;
        factor_diff_matrix[idx_t] /= normalizer;
    }
}

template <typename Dtype>
void InflationXLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    Dtype* factor_;
    if (iter_counter_ >= this->layer_param().inflation_factor_param().start_iter())
    	factor_ = bottom[1]->mutable_cpu_data();
    else {
	factor_ = new Dtype();
	*factor_ = this->layer_param().inflation_factor_param().factor();
    }

    // get parameters
    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    Dtype* factor_diff_matrix = factor_diff_.mutable_gpu_data();
    const Dtype* label = NULL;
    if (this->layer_param().inflation_factor_param().use_bg_mask() == true)
	label = bottom[1]->gpu_data();
    
    // new shape
    const int top_height = top[0]->height();
    const int top_width = top[0]->width();

    // resize
    const int nthreads = top_height * top_width;
    for (int n = 0; n < num; n++) {
        for (int c = 0; c < channels; c++) {    
      
            const int index_in = (n * channels + c) * height * width;
            const int index_out = (n * channels + c) * top_height * top_width;

	    if (this->layer_param().inflation_factor_param().use_bg_mask() == true)
	    {
		const int index_label = n * bottom[1]->height() * bottom[1]->width();
                InflateForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data + index_in, height, width, top_data + index_out, top_height, top_width, *factor_, factor_diff_matrix, margin_, this->factor_bg_mask, this->bg_mask_weight, label+index_label);
	    }
	    else
                InflateForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data + index_in, height, width, top_data + index_out, top_height, top_width, *factor_, factor_diff_matrix, margin_);
        }
    }   
}

template <typename Dtype>
__global__ void InflateBackwardGPU(const int nthreads, 
            Dtype *bottom_diff, const int bottom_height, const int bottom_width, 
            const Dtype *top_diff, const int top_height, const int top_width, 
            const float factor, const int margin) {

    const float normalizer = factor * factor * margin * margin * margin * margin;

    CUDA_KERNEL_LOOP(index, nthreads) {
        
        // index refers to to top_data
        const int n = index / bottom_width;
        const int m = index % bottom_width;
        
        const int idx_s = n * bottom_width + m;
        bottom_diff[idx_s] = 0;
        
        for (int y_t = MAX(floor((n - margin) * factor) + 1, 0); y_t < MIN((n + margin) * factor, top_height); y_t++) {
            for (int x_t = MAX(floor((m - margin) * factor) + 1, 0); x_t < MIN((m + margin) * factor, top_width); x_t++) {
                
                // diff
                bottom_diff[idx_s] += top_diff[y_t * top_width + x_t] 
                                      * (margin - abs((x_t / factor) - m)) * (margin - abs((y_t / factor) - n));
            }
        }
        bottom_diff[idx_s] /= normalizer;
    }
}

template <typename Dtype>
void InflationXLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    

    // get parameters
    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();
    const int top_height = top[0]->height();
    const int top_width = top[0]->width();
    
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* factor_;
    if (iter_counter_ >= this->layer_param().inflation_factor_param().start_iter())
    	factor_ = bottom[1]->mutable_cpu_data();
    else {
	factor_ = new Dtype();
	*factor_ = this->layer_param().inflation_factor_param().factor();
    }
    Dtype* factor_diff = bottom[1]->mutable_cpu_diff();
    const Dtype* factor_diff_matrix = factor_diff_.cpu_data();

    if (propagate_down[0]) {

        // compute diff for bottom
        const int nthreads = height * width;
        
        for (int n = 0; n < num; n++) {
            for (int c = 0; c < channels; c++) {
                const int index_in = (n * channels + c) * height * width;
                const int index_out = (n * channels + c) * top_height * top_width;
                InflateBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_diff + index_in, height, width, top_diff + index_out, top_height, top_width, *factor_, margin_);
            }
        }
    }
    
    if (iter_counter_ >= this->layer_param().inflation_factor_param().start_iter()) {
    
	if (iter_counter_ % 20 == 0)    *factor_diff = 0;

        // compute diff for factor_
        // dL/d(factor) = sum(top.diff[i,j] * d(top.data[i,j])/d(factor))
        Dtype sum_dLoss_dfactor = caffe_cpu_dot(top[0]->count(), factor_diff_matrix, top[0]->cpu_diff());

        const Dtype* top_factor_diff = top[1]->cpu_diff();       
        
	Dtype tmp = static_cast<Dtype>(1.0 * sum_dLoss_dfactor / num / height / width + top_factor_diff[0]);
	tmp *= this->layer_param().inflation_factor_param().factor_diff_weight();

	if (this->layer_param().inflation_factor_param().clip_gradient() == true)
	{
	    float MARGIN = this->layer_param().inflation_factor_param().clip_gradient_value();
	    if (tmp > MARGIN) tmp = MARGIN;
	    if (tmp < -MARGIN) tmp = -MARGIN;
	}
        *factor_diff += tmp;

        LOG(INFO) << " No." << iter_counter_ % 20
                  << "  factor: " << *factor_
                  << "  (" << height << " -> " << top_height << ")"
                  << "  f_diff: " << sum_dLoss_dfactor / num / height / width
		  << "  diff: " << tmp
		  << " Total diff: " << *factor_diff;
	if (iter_counter_ % 20 == 19)    LOG(INFO) << " Total diff: " << *factor_diff;

    } else {
        *factor_diff = 0;
        
        if (iter_counter_ == this->layer_param().inflation_factor_param().start_iter() && propagate_down[0])
            LOG(INFO) << " Start learning factor value";    
    }
    iter_counter_++;
}

INSTANTIATE_LAYER_GPU_FUNCS(InflationXLayer);

}  // namespace caffe
