#include <vector>

#include "caffe/layers/base_data_layer.hpp"
<<<<<<< HEAD
#include "caffe/layers/image_seg_data_layer.hpp"
=======
>>>>>>> caffe-bvlc-dev/master

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
<<<<<<< HEAD
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void ImageDimPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, join the thread
  BasePrefetchingDataLayer<Dtype>::JoinPrefetchThread();
  // Copy the data
  caffe_copy(this->prefetch_data_.count(), this->prefetch_data_.gpu_data(),
	     top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    caffe_copy(this->prefetch_label_.count(), this->prefetch_label_.gpu_data(),
	       top[1]->mutable_gpu_data());
  }
  if (output_data_dim_) {
    caffe_copy(prefetch_data_dim_.count(), prefetch_data_dim_.gpu_data(),
	       top[2]->mutable_gpu_data());
  }

  // Start a new prefetch thread
  BasePrefetchingDataLayer<Dtype>::CreatePrefetchThread();
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(ImageDimPrefetchingDataLayer);
=======
  if (prefetch_current_) {
    prefetch_free_.push(prefetch_current_);
  }
  prefetch_current_ = prefetch_full_.pop("Waiting for data");
  // Reshape to loaded data.
  top[0]->ReshapeLike(prefetch_current_->data_);
  top[0]->set_gpu_data(prefetch_current_->data_.mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(prefetch_current_->label_);
    top[1]->set_gpu_data(prefetch_current_->label_.mutable_gpu_data());
  }
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
>>>>>>> caffe-bvlc-dev/master

}  // namespace caffe
