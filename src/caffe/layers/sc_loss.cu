#include "hip/hip_runtime.h"
/*
* Statistical contextual loss layer for segmentation.
* Author: Wei Zhen @ CS, HUST
* Create on: 2017-10-24
* Last Modified: 2017-10-24
*/

#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/sc_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Compute_distance_data_gpu(int nthreads,
		 const Dtype* bottom_label,
		 Dtype* distance_data,
		 const Dtype* bottom_data, 
		 const Dtype* center, 
		 const int dim, const int c,
		 bool is_param_propagate_down_,
		 Dtype* variation_sum_data,
		 int* label_counter_,
		 float label_bottom_factor,
		 const int label_width, const int data_width,
		 const int* ignore_label, const int ignore_label_size,
		 Dtype* hw_flags) {
	CUDA_KERNEL_LOOP(index, nthreads) {
	    // convert data idx to label idx
	    const int y = int(index / data_width);
	    const int x = index % data_width;
	    const int y_ = int(y * label_bottom_factor);
	    const int x_ = int(x * label_bottom_factor);
	    const int label_idx = y_*label_width + x_;
            const int label_value = static_cast<int>(bottom_label[label_idx]);
	    // ignore label
	    bool ignore_label_flag = false;
	    for (int i = 0; i < ignore_label_size; i++) {
		if (ignore_label[i] == label_value)
		    ignore_label_flag = true;
	    }
	    if (ignore_label_flag)    continue;
            // D(n,c,y,x) = X(n,c,y,x) - C(c,Y(n,1,y,x))
            distance_data[index] = bottom_data[index] - center[label_value*dim + c];
	    // compute center diff
	    if (is_param_propagate_down_) {
		// variation_sum_data(Y(n,1,y,x), c) -= D(n,c,x,y) + 2x center_mutual_distance{finished in backward}
		if (hw_flags != NULL) {								// hard awared mode
			if (hw_flags[index] > 0) {						// flag>0 => true; flag<0 => false
			    variation_sum_data[label_value*dim + c] -= distance_data[index];
			    label_counter_[label_value]++;
			}
		}
		else {
			variation_sum_data[label_value*dim + c] -= distance_data[index];
			label_counter_[label_value]++;
		}
	    }
        }
}

template <typename Dtype>
__global__ void Compute_inter_loss_term_gpu(int nthreads,
		 const Dtype* bottom_label,
		 Dtype* distance_data,
		 const int dim, const int c,
		 const Dtype* distance_inter,
		 int* label_counter_,
		 float label_bottom_factor,
		 const int label_width, const int data_width,
		 const int* ignore_label, const int ignore_label_size,
		 const float lambda_, const bool is_hard_aware_,
		 Dtype* hw_flags) {
	CUDA_KERNEL_LOOP(index, nthreads) {
	    // convert data idx to label idx
	    const int y = int(index / data_width);
	    const int x = index % data_width;
	    const int y_ = int(y * label_bottom_factor);
	    const int x_ = int(x * label_bottom_factor);
	    const int label_idx = y_*label_width + x_;
            const int label_value = static_cast<int>(bottom_label[label_idx]);
	    // ignore label
	    bool ignore_label_flag = false;
	    for (int i = 0; i < ignore_label_size; i++) {
		if (ignore_label[i] == label_value)
		    ignore_label_flag = true;
	    }
	    if (ignore_label_flag)    continue;
	    // if don't use hard aware mode or use hard aware mode and the flag>0,
	    //    then propagate down inter loss diff
	    if (is_hard_aware_==false || (is_hard_aware_==true && hw_flags[index]>0)) {
		distance_data[index] += 2 * lambda_ / label_counter_[label_value] * distance_inter[label_value*dim+c];
	    }
	    // else, if use hard aware mode and the flag<=0, then do nothing
	    else {}
        }
}

template <typename Dtype>
void StatisticContextualLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int num = bottom[0]->num();
  const int dim = bottom[0]->channels();
  const int label_height = bottom[1]->height();
  const int label_width = bottom[1]->width();
  const int data_width = bottom[0]->width();

  /*
  * 1. inter center distances
  */
  // accumulate all mutual center distances
  Dtype* tmp_sub = (Dtype*)malloc(dim*sizeof(Dtype));
  Dtype* distance_inter = center_mutual_distance.mutable_cpu_data();
  // reset mutual center distances
  caffe_set(center_mutual_distance.count(), (Dtype)0., distance_inter);
  const Dtype* center = this->blobs_[0]->cpu_data();
  for (int i = 0; i < label_num_; ++i) {
    if (find(ignore_label_.begin(), ignore_label_.end(), i) != ignore_label_.end())  continue;
    caffe_set(dim, (Dtype)0., tmp_sub);
    for (int j = 0; j < label_num_; ++j) {
	if (find(ignore_label_.begin(), ignore_label_.end(), j) != ignore_label_.end())  continue;
	if (i == j)  continue;
	// |current center (i,i) - another center (j,i)|^2, i != j
	// |current center (i,j) - another center (j,j)|^2, i != j
	tmp_sub[i] = center[i*dim + i] - center[j*dim + i];
	tmp_sub[j] = center[i*dim + j] - center[j*dim + j];
	caffe_axpy(dim, (Dtype)1., tmp_sub, distance_inter+i*dim);
    }
    distance_inter[i*dim + i] /= (label_num_ - ignore_label_.size());
  // L_{D} = max(ld_margin_ - center_mutual_distance^2, 0)
  if (caffe_cpu_dot(dim, distance_inter+i*dim, distance_inter+i*dim) > this->ld_margin_)
    caffe_set(dim, (Dtype)0., distance_inter+i*dim);
  }
  free(tmp_sub);
  
  /*
  * 2. class intra distances and intra term diffs
  */
  /* 2.1 copy var into gpu space */
  // convert ignore label vector into array
  int* cu_ignore_label = NULL;
  hipMalloc((void**)&cu_ignore_label, this->ignore_label_.size() * sizeof(int));
  // NOTE!!! first param is number of byte, not number of a data
  caffe_gpu_memcpy(this->ignore_label_.size() * sizeof(int), &this->ignore_label_[0], cu_ignore_label);

  // prepare hard aware flags
  Dtype* hw_flags = NULL;
  if (bottom.size() == 3 && this->is_hard_aware_ == true) {	// hard awared mode
	hw_flags = this->hard_aware_flags_.mutable_cpu_data();
  }

  // the i-th distance_data
  for (int n = 0; n < num; ++n) {
    for (int c = 0; c < dim; ++c) {
	const int c_idx = n*dim+c;
	const Dtype* bottom_label = bottom[1]->gpu_data()+n*label_height*label_width;
	const Dtype* bottom_data = bottom[0]->gpu_data()+c_idx*inner_num_;
	Dtype* distance_data = distance_.mutable_gpu_data() + c_idx*inner_num_;
	Compute_distance_data_gpu<Dtype><<<CAFFE_GET_BLOCKS(inner_num_), CAFFE_CUDA_NUM_THREADS>>>
		(inner_num_,					//nthreads
		 bottom_label,					//bottom_label
		 distance_data,					//distance_data
		 bottom_data,					//bottom_data
		 this->blobs_[0]->gpu_data(), dim, c,		//center, dim, c
		 this->param_propagate_down_[0], 		//is_param_propagate_down_
		 variation_sum_.mutable_gpu_data(),		//variation_sum_data
		 label_counter_.mutable_gpu_data(),		//label_counter_
		 this->label_bottom_factor,			//label_bottom_factor
		 label_width, data_width,			//label_width, data_width
		 cu_ignore_label, this->ignore_label_.size(),   //ignore_label, ignore_label_size
		 hw_flags);					//hard aware flags
    }
  }
  for (int i = 0; i < label_counter_.count(); i++)
    label_counter_.mutable_cpu_data()[i] /= dim;

  /*
  * 3. inter diffs
  */
  for (int n = 0; n < num; ++n) {
    for (int c = 0; c < dim; ++c) {
	const int c_idx = n*dim+c;
	const Dtype* bottom_label = bottom[1]->gpu_data()+n*label_height*label_width;
	const Dtype* bottom_data = bottom[0]->gpu_data()+c_idx*inner_num_;
	Dtype* distance_data = distance_.mutable_gpu_data() + c_idx*inner_num_;
	Compute_inter_loss_term_gpu<Dtype><<<CAFFE_GET_BLOCKS(inner_num_), CAFFE_CUDA_NUM_THREADS>>>
		(inner_num_,					//nthreads
		 bottom_label,					//bottom_label
		 distance_data,					//distance_data
		 dim, c,					//dim, c
		 center_mutual_distance.gpu_data(),		//distance_inter
		 label_counter_.mutable_gpu_data(),		//label_counter_
		 this->label_bottom_factor,			//label_bottom_factor
		 label_width, data_width,			//label_width, data_width
		 cu_ignore_label, this->ignore_label_.size(),   //ignore_label, ignore_label_size
		 this->lambda_, this->is_hard_aware_,		//lambda_, is_hard_aware_
		 hw_flags);
    }
  }

  /*
  * 4. compute loss
  */
  Dtype loss = caffe_cpu_dot(distance_.count(), distance_.cpu_data(), distance_.cpu_data());
  loss = loss / num / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;

  hipFree(cu_ignore_label);
}

template <typename Dtype>
void StatisticContextualLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  /*
  * 1. Gradient with respect to centers
  */
  if (this->param_propagate_down_[0]) {
    Dtype* center_diff = this->blobs_[0]->mutable_cpu_diff();
    const Dtype* center = this->blobs_[0]->mutable_cpu_data();
    const Dtype* variation_sum_data = variation_sum_.cpu_data();
    const int* label_counter__ = label_counter_.cpu_data();
    const int dim = bottom[0]->channels();

    caffe_set(this->blobs_[0]->count(), (Dtype)0., center_diff);
    /*
    * 1.1 center's diff from other centers
    */
    // lambda is the balance weight between two different gradients
    caffe_axpy(dim*label_num_, this->lambda_*2, center_mutual_distance.cpu_data(), center_diff);
    /*
    * 1.2 center's diff from the cluster itself
    */
    for (int label_value = 0; label_value < label_num_; label_value++) {
      // ignore label
      if (find(ignore_label_.begin(), ignore_label_.end(), label_value) != ignore_label_.end())  continue;
      caffe_axpy(dim, (Dtype)1/(label_counter__[label_value] + (Dtype)1.), variation_sum_data + label_value*dim, center_diff + label_value*dim);
      /*
      * 1.3 center decay, on non-nagative values on axises of other labels
      */
      for (int i = 0; i < dim; i++) {
	if ((i != label_value) && (center[label_value*dim+i] > 0))
	    center_diff[label_value*dim + i] = (Dtype)this->center_decay_ * center[label_value*dim+i];
      }
    }

/*Dtype a=0, b=0, c=0;
for (int i = 0; i < dim; i++){
a+=center_mutual_distance.cpu_data()[1*dim+i]*this->lambda_;
b+=variation_sum_data[1*dim+i]/(label_counter__[1] + (Dtype)1.)*bottom[0]->channels();
c+=center_diff[1*dim+i];
}
const Dtype* test_val = center_mutual_distance.cpu_data()+(1*dim);
printf("%f %f %f %f\n",a,b,c, caffe_cpu_dot(dim, test_val, test_val));*/

    // reset variation_sum_
    caffe_set(variation_sum_.count(), (Dtype)0., variation_sum_.mutable_cpu_data());
    // reset label counter
    caffe_set(label_counter_.count(), (int)0., label_counter_.mutable_cpu_data());

//LOG(INFO)<<"!!"<<center[0]<<" "<<center[1]<<" "<<center[2]<<" "<<center[3]<<" "<<center[4]<<" "<<center[5]<<" "<<center[6];
  }

  /*
  * 2. Gradient with respect to bottom data (they are acutually computed in forward func)
  */
  // Gradient with respect to bottom data 
  if (propagate_down[0]) {
    caffe_cpu_scale(distance_.count(), top[0]->cpu_diff()[0] / bottom[0]->num(), distance_.cpu_data(), bottom[0]->mutable_cpu_diff());
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(StatisticContextualLossLayer);

} // namespace caffe
