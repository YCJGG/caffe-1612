#include "hip/hip_runtime.h"
/*
* Modified center loss layer for segmentation.
* Author: Wei Zhen @ IIE, CAS
* Create on: 2016-12-25
* Last Modified: 2016-02-25
*/

#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/center_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Compute_distance_data_gpu(int nthreads,
		 const Dtype* bottom_label,
		 Dtype* distance_data,
		 const Dtype* bottom_data, 
		 const Dtype* center, 
		 const int dim, const int c,
		 bool is_param_propagate_down_,
		 Dtype* variation_sum_data,
		 int* label_counter_,
		 float label_bottom_factor,
		 const int label_width, const int data_width,
		 const int* ignore_label, const int ignore_label_size) {
	CUDA_KERNEL_LOOP(index, nthreads) {
	    // convert data idx to label idx
	    const int y = int(index / data_width);
	    const int x = index % data_width;
	    const int y_ = int(y * label_bottom_factor);
	    const int x_ = int(x * label_bottom_factor);
	    const int label_idx = y_*label_width + x_;
            const int label_value = static_cast<int>(bottom_label[label_idx]);
	    // ignore label
	    bool ignore_label_flag = false;
	    for (int i = 0; i < ignore_label_size; i++) {
		if (ignore_label[i] == label_value)
		    ignore_label_flag = true;
	    }
	    if (ignore_label_flag)    continue;
            // D(n,c,y,x) = X(n,c,y,x) - C(c,Y(n,1,y,x))
            distance_data[index] = bottom_data[index] - center[label_value*dim + c];
	    // compute center diff
	    if (is_param_propagate_down_) {
		// variation_sum_data(Y(n,1,y,x), c) -= D(n,c,x,y) + 2x center_mutual_distance{finished in backward}
		variation_sum_data[label_value*dim + c] -= distance_data[index];
		label_counter_[label_value]++;
	    }
        }
}

template <typename Dtype>
void CenterLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int num = bottom[0]->num();
  const int dim = bottom[0]->channels();
  const int label_height = bottom[1]->height();
  const int label_width = bottom[1]->width();
  const int data_width = bottom[0]->width();

/*
  // find shortest center distance for each center
  Dtype tmp_distance = 1e20;
  Dtype* tmp_sub = (Dtype*)malloc(dim*sizeof(Dtype));
  Dtype* distance_inter = center_mutual_distance.mutable_cpu_data();
  const Dtype* center = this->blobs_[0]->cpu_data();
  for (int i = 0; i < label_num_; ++i) {
    if (find(ignore_label_.begin(), ignore_label_.end(), i) != ignore_label_.end())  continue;
    for (int j = 0; j < label_num_; ++j) {
	if (find(ignore_label_.begin(), ignore_label_.end(), j) != ignore_label_.end())  continue;
	if (i == j)  continue;
	// |current center (i) - another center (j)|^2, i != j
	caffe_sub(dim, center+i*dim, center+j*dim, tmp_sub);
	Dtype tmp = caffe_cpu_dot(dim, tmp_sub, tmp_sub);
	if (tmp < tmp_distance) {
	    tmp_distance = tmp;
	    caffe_copy(dim, tmp_sub, distance_inter+i*dim);
	}
    }
  }
*/
  // accumulate all mutual center distances
  Dtype* tmp_sub = (Dtype*)malloc(dim*sizeof(Dtype));
  Dtype* distance_inter = center_mutual_distance.mutable_cpu_data();
  // reset mutual center distances
  caffe_set(center_mutual_distance.count(), (Dtype)0., distance_inter);
  const Dtype* center = this->blobs_[0]->cpu_data();
  for (int i = 0; i < label_num_; ++i) {
    if (find(ignore_label_.begin(), ignore_label_.end(), i) != ignore_label_.end())  continue;
    for (int j = 0; j < label_num_; ++j) {
	if (find(ignore_label_.begin(), ignore_label_.end(), j) != ignore_label_.end())  continue;
	if (i == j)  continue;
	// |current center (i) - another center (j)|^2, i != j
	caffe_sub(dim, center+i*dim, center+j*dim, tmp_sub);
	caffe_axpy(dim, (Dtype)1./label_num_, tmp_sub, distance_inter+i*dim);
    }
  }
  
  // convert ignore label vector into array
  int* cu_ignore_label = NULL;
  hipMalloc((void**)&cu_ignore_label, this->ignore_label_.size() * sizeof(int));
  // NOTE!!! first param is number of byte, not number of a data
  caffe_gpu_memcpy(this->ignore_label_.size() * sizeof(int), &this->ignore_label_[0], cu_ignore_label);

  // the i-th distance_data
  for (int n = 0; n < num; ++n) {
    for (int c = 0; c < dim; ++c) {
	const int c_idx = n*dim+c;
	const Dtype* bottom_label = bottom[1]->gpu_data()+n*label_height*label_width;
	const Dtype* bottom_data = bottom[0]->gpu_data()+c_idx*inner_num_;
	Dtype* distance_data = distance_.mutable_gpu_data() + c_idx*inner_num_;
	Compute_distance_data_gpu<Dtype><<<CAFFE_GET_BLOCKS(inner_num_), CAFFE_CUDA_NUM_THREADS>>>
		(inner_num_,					//nthreads
		 bottom_label,					//bottom_label
		 distance_data,					//distance_data
		 bottom_data,					//bottom_data
		 this->blobs_[0]->gpu_data(), dim, c,		//center, dim, c
		 this->param_propagate_down_[0], 		//is_param_propagate_down_
		 variation_sum_.mutable_gpu_data(),		//variation_sum_data
		 label_counter_.mutable_gpu_data(),		//label_counter_
		 this->label_bottom_factor,			//label_bottom_factor
		 label_width, data_width,			//label_width, data_width
		 cu_ignore_label, this->ignore_label_.size());   //ignore_label, ignore_label_size
    }
  }

  // compute loss
  Dtype loss = caffe_cpu_dot(distance_.count(), distance_.cpu_data(), distance_.cpu_data());
  loss = loss / num / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;

  hipFree(cu_ignore_label);
  free(tmp_sub);
}

template <typename Dtype>
void CenterLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  // Gradient with respect to centers
  if (this->param_propagate_down_[0]) {
    Dtype* center_diff = this->blobs_[0]->mutable_cpu_diff();
    const Dtype* variation_sum_data = variation_sum_.cpu_data();
    const int* label_counter__ = label_counter_.cpu_data();
    const int dim = bottom[0]->channels();

    // center's diff from other centers, update after late_iter_
    if (count_ > this->late_iter_) {
	caffe_set(this->blobs_[0]->count(), (Dtype)0., center_diff);
	// second input is the balance weight between two different gradients
	caffe_axpy(dim*label_num_, (Dtype)0.1, center_mutual_distance.cpu_data(), center_diff);
    }

    for (int label_value = 0; label_value < label_num_; label_value++) {
      // ignore label
      if (find(ignore_label_.begin(), ignore_label_.end(), label_value) != ignore_label_.end())  continue;
      caffe_axpy(dim, (Dtype)1./(label_counter__[label_value] + (Dtype)1.), variation_sum_data + label_value*dim, center_diff + label_value*dim);
    }

//Dtype a=0, b=0, c=0;
//for (int i = 0; i < dim; i++){
//a+=center_mutual_distance.cpu_data()[1*dim+i];
//b+=variation_sum_data[1*dim+i];
//c+=center_diff[1*dim+i];
//}
//printf("%f %f %f\n",a,b,c);


    // reset variation_sum_
    caffe_set(variation_sum_.count(), (Dtype)0., variation_sum_.mutable_cpu_data());
    // reset label counter
    caffe_set(label_counter_.count(), (int)0., label_counter_.mutable_cpu_data());
  }
  // Gradient with respect to bottom data 
  if (propagate_down[0] && count_ > this->late_iter_) {
    caffe_cpu_scale(distance_.count(), top[0]->cpu_diff()[0] / bottom[0]->num(), distance_.cpu_data(), bottom[0]->mutable_cpu_diff());
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CenterLossLayer);

} // namespace caffe
