#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/crop_layer.hpp"

namespace caffe {

<<<<<<< HEAD
// Copy (one line per thread) from one array to another, with arbitrary
// strides in the last two dimensions.
template <typename Dtype>
__global__ void copy_kernel(const int n, const int height, const int width,
    const int src_outer_stride, const int src_inner_stride,
    const int dest_outer_stride, const int dest_inner_stride,
    const Dtype* src, Dtype* dest) {
  CUDA_KERNEL_LOOP(index, n) {
    int src_start = index / height * src_outer_stride
                  + index % height * src_inner_stride;
    int dest_start = index / height * dest_outer_stride
                   + index % height * dest_inner_stride;
    for (int i = 0; i < width; ++i) {
      dest[dest_start + i] = src[src_start + i];
    }
=======
__device__ int compute_uncropped_index(
    int index,
    const int ndims,
    const int* src_strides,
    const int* dest_strides,
    const int* offsets) {
  int dest_index = index;
  int src_index = 0;
  for (int i = 0; i < ndims; ++i) {
      int coord = dest_index / dest_strides[i];
      dest_index -= coord * dest_strides[i];
      src_index += src_strides[i] * (coord + offsets[i]);
  }
  return src_index;
}

template <typename Dtype>
__global__ void crop_kernel_forward(const int nthreads,
    const int ndims,
    const int* src_strides,
    const int* dest_strides,
    const int* offsets,
    const Dtype* src, Dtype* dest) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int src_index = compute_uncropped_index(
        index, ndims, src_strides, dest_strides, offsets);
    dest[index] = src[src_index];
>>>>>>> caffe-bvlc-dev/master
  }
}

template <typename Dtype>
<<<<<<< HEAD
void CropLayer<Dtype>::crop_copy_gpu(const vector<Blob<Dtype>*>& bottom,
             const vector<Blob<Dtype>*>& top,
             const vector<int>& offsets,
             vector<int> indices,
             int cur_dim,
             const Dtype* src_data,
             Dtype* dest_data,
             bool is_forward) {
  if (cur_dim + 2 < top[0]->num_axes()) {
    // We are not yet at the final dimension, call copy recursivley
    for (int i = 0; i < top[0]->shape(cur_dim); ++i) {
      indices[cur_dim] = i;
      crop_copy_gpu(bottom, top, offsets, indices, cur_dim+1,
                src_data, dest_data, is_forward);
    }
  } else {
    // We are at the last two dimensions, which are stored continuously in
    // memory. With (N,C,H,W)
    //              (0,1,2,3) cur_dim   -> H
    //                        cur_dim+1 -> W
    const int lines = top[0]->shape(cur_dim);
    const int height = top[0]->shape(cur_dim);
    const int width = top[0]->shape(cur_dim+1);
    std::vector<int> ind_off(cur_dim+2, 0);
    for (int j = 0; j < cur_dim; ++j) {
        ind_off[j] = indices[j] + offsets[j];
    }
    ind_off[cur_dim] = offsets[cur_dim];
    ind_off[cur_dim+1] = offsets[cur_dim+1];
    // Compute copy strides
    const int src_outer_stride =
        bottom[0]->shape(cur_dim)*bottom[0]->shape(cur_dim+1);
    const int src_inner_stride = bottom[0]->shape(cur_dim+1);
    const int dest_outer_stride =
        top[0]->shape(cur_dim)*top[0]->shape(cur_dim+1);
    const int dest_inner_stride = top[0]->shape(cur_dim+1);

    if (is_forward) {
      const Dtype* bottom_data = bottom[0]->gpu_data() +
          bottom[0]->offset(ind_off);
      Dtype* top_data = top[0]->mutable_gpu_data() +
          top[0]->offset(indices);
      // NOLINT_NEXT_LINE(whitespace/operators)
      copy_kernel<<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(
          lines, height, width,
          src_outer_stride, src_inner_stride,
          dest_outer_stride, dest_inner_stride,
          bottom_data, top_data);

    } else {
      const Dtype* top_diff = top[0]->gpu_diff() +
          top[0]->offset(indices);
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff() +
          bottom[0]->offset(ind_off);
      // NOLINT_NEXT_LINE(whitespace/operators)
      copy_kernel<<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(
          lines, height, width,
          dest_outer_stride, dest_inner_stride,
          src_outer_stride, src_inner_stride,
          top_diff, bottom_diff);
    }
=======
__global__ void crop_kernel_backward(const int nthreads,
    const int ndims,
    const int* src_strides,
    const int* dest_strides,
    const int* offsets,
    Dtype* src, const Dtype* dest) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int src_index = compute_uncropped_index(
        index, ndims, src_strides, dest_strides, offsets);
    src[src_index] = dest[index];
>>>>>>> caffe-bvlc-dev/master
  }
}

template <typename Dtype>
void CropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
<<<<<<< HEAD
  std::vector<int> indices(top[0]->num_axes(), 0);
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  crop_copy_gpu(bottom, top, offsets, indices, 0, bottom_data, top_data, true);
=======
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int n = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  crop_kernel_forward<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n,
      bottom[0]->num_axes(),
      src_strides_.gpu_data(),
      dest_strides_.gpu_data(),
      offsets.gpu_data(),
      bottom_data, top_data);
>>>>>>> caffe-bvlc-dev/master
}

template <typename Dtype>
void CropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
<<<<<<< HEAD

  if (propagate_down[0]) {
    caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
    std::vector<int> indices(top[0]->num_axes(), 0);
    crop_copy_gpu(bottom, top, offsets, indices, 0, top_diff, bottom_diff,
                  false);
=======
  int n = top[0]->count();

  if (propagate_down[0]) {
    caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
    // NOLINT_NEXT_LINE(whitespace/operators)
    crop_kernel_backward<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n,
        bottom[0]->num_axes(),
        src_strides_.gpu_data(),
        dest_strides_.gpu_data(),
        offsets.gpu_data(),
        bottom_diff, top_diff);
>>>>>>> caffe-bvlc-dev/master
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CropLayer);

}  // namespace caffe
